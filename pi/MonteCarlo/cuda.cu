// Written by Barry Wilkinson, UNC-Charlotte. Pi.cu  December 22, 2010.
//Derived somewhat from code developed by Patrick Rogers, UNC-C

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

#define TRIALS_PER_THREAD 4096
#define BLOCKS 256
#define THREADS 256
#define PI 3.141592654  // known value of pi

__global__ void gpu_monte_carlo(float *estimate, hiprandState *states) {
	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int points_in_circle = 0;
	float x, y;

	hiprand_init(142857, tid, 0, &states[tid]);  // 	Initialize CURAND


	for(int i = 0; i < TRIALS_PER_THREAD; i++) {
		x = hiprand_uniform (&states[tid]);
		y = hiprand_uniform (&states[tid]);
		points_in_circle += (x*x + y*y <= 1.0f); // count if x & y is in the circle.
	}
	estimate[tid] = 4.0f * points_in_circle / (float) TRIALS_PER_THREAD; // return estimate of pi
}

int main (int argc, char *argv[]) {
	clock_t start, stop;
	float host[BLOCKS * THREADS];
	float *dev;
	hiprandState *devStates;

	printf("# of trials per thread = %d, # of blocks = %d, # of threads/block = %d.\n", TRIALS_PER_THREAD,
BLOCKS, THREADS);

	start = clock();

	hipMalloc((void **) &dev, BLOCKS * THREADS * sizeof(float)); // allocate device mem. for counts
	
	hipMalloc( (void **)&devStates, THREADS * BLOCKS * sizeof(hiprandState) );

	gpu_monte_carlo<<<BLOCKS, THREADS>>>(dev, devStates);

	hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(float), hipMemcpyDeviceToHost); // return results 

	float pi_gpu;
	for(int i = 0; i < BLOCKS * THREADS; i++) {
		pi_gpu += host[i];
	}

	pi_gpu /= (BLOCKS * THREADS);

	stop = clock();

	printf("GPU pi calculated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC);
	printf("CUDA estimate of PI = %f [error of %f]\n", pi_gpu, pi_gpu - PI);
	
	return 0;
}
